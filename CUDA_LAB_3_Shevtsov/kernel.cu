﻿
#include "hip/hip_runtime.h"

#include <hip/device_functions.h>

#include <iomanip>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <stdio.h>


void GenerateRandomMatrix(int* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = 0;
        }
        if (rand() % 2 == 0) {
            int random_col = rand() % cols;
            matrix[i * cols + random_col] = 1;
        }
    }
}


void PrintMatrix(int* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

// Функция на CPU для проверки симметрии строк относительно средней вертикальной линии матрицы
void checkSymmetryCPU(const int* matrix, bool* result, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        result[i] = true;
        for (int j = 0; j < cols / 2; j++) {
            if (matrix[i * cols + j] != matrix[i * cols + (cols - 1 - j)]) {
                result[i] = false;  
            }
        }
    }
}

// Функция на GPU для проверки симметрии строк относительно средней вертикальной линии матрицы
__global__ void _IsRowSymmetricGPU(const int* matrix, bool* result, int rows, int cols) {
    int rowIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowIdx < rows) {
        const int* row = matrix + rowIdx * cols;
        result[rowIdx] = true;
        for (int i = 0; i < cols / 2; ++i) {
            if (row[i] != row[cols - i - 1]) {
                result[rowIdx] = false;
            }
        }
    }
}

// Оптимизированная функция на GPU для проверки симметрии строк относительно средней вертикальной линии матрицы
__global__ void checkSymmetryGPU(int* matrix, bool* result, int rows, int cols) {
    __shared__ int sharedCache[256][32]; // Кеш в разделяемой памяти для 256 строк, каждая строка - 16 элементов слева + 16 элементов справа

    const int t = threadIdx.x;
    const int bx = blockIdx.x;
    bool isSymmetric = true;
    for (int part = 0; part < cols / 32; part++) {
        // Загружаем 16 элементов слева и 16 элементов справа в кеш для каждой из 256 строк
        for (int k = 0; k < 16; k++) {

            sharedCache[t / 16 + k * 16][t % 16] = matrix[(bx * blockDim.x + t / 16 + k * 16) * cols + part * 16 + t % 16];
            sharedCache[t / 16 + k * 16][32 - 1 - t % 16] = matrix[(bx * blockDim.x + t / 16 + k * 16) * cols + cols - 1 - part * 16 - t % 16];
        }

        // Барьер синхронизации для ожидания, пока все потоки загрузят кеш
        __syncthreads();

        // Проверяем симметрию для текущей строки
        for (int j = 0; j < 32 / 2; j++) {
            if (sharedCache[t][j] != sharedCache[t][32 - 1 - j]) {
                isSymmetric = false; 
            }
        }
        __syncthreads();
    }
    result[bx * blockDim.x + t] = isSymmetric;
}


int main() {
    hipEvent_t startCUDA, stopCUDA;
    clock_t startCPU;
    float elapsedTimeCUDA, elapsedTimeCPU;

    hipEventCreate(&startCUDA);
    hipEventCreate(&stopCUDA);

    srand(static_cast<unsigned>(time(nullptr))); // Инициализируем генератор случайных чисел

    const int rows = 50000;
    const int cols = 1024;
    const int matrixSize = rows * cols;

    const int blockDim = 256;
    const int numBlocks = (rows + 256 - 1) / 256;

    int* h_matrix = new int[matrixSize]; 
    bool* h_symmetryResults = new bool[rows];
    bool* cpu_symmetryResults = new bool[rows];

    // Генерируем случайную матрицу на хосте
    GenerateRandomMatrix(h_matrix, rows, cols);

    startCPU = clock();

    // Проверяем симметрию строк и сохраняем результаты в векторе
    checkSymmetryCPU(h_matrix, cpu_symmetryResults, rows, cols);
    
    elapsedTimeCPU = (float)(clock() - startCPU) / CLOCKS_PER_SEC;

    // Выделяем память на устройстве
    int* d_matrix;
    bool* d_symmetryResults;
    hipMalloc((void**)&d_matrix, sizeof(int) * matrixSize);
    hipMalloc((void**)&d_symmetryResults, sizeof(bool) * rows);

    // Копируем матрицу с хоста на устройство
    hipMemcpy(d_matrix, h_matrix, sizeof(int) * matrixSize, hipMemcpyHostToDevice);

    hipEventRecord(startCUDA, 0);
   
    // Вызываем на GPU для проверки симметрии
    checkSymmetryGPU <<<numBlocks, blockDim>>> (d_matrix, d_symmetryResults, rows, cols);

    hipEventRecord(stopCUDA, 0);
    hipEventSynchronize(stopCUDA);
    hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

    // Копируем результаты с устройства на хост
    hipMemcpy(h_symmetryResults, d_symmetryResults, sizeof(bool) * rows, hipMemcpyDeviceToHost);

    // Освобождаем память на устройстве
    hipFree(d_matrix);
    hipFree(d_symmetryResults);

    // Выводим результаты на консоль
    //for (int i = 0; i < rows; ++i) {
    //    std::cout << "Row " << i << ": " << (h_symmetryResults[i] ? "Symmetric" : "Not Symmetric") << "\t"
    //         << (cpu_symmetryResults[i] ? "Symmetric" : "Not Symmetric") << std::endl;
    //}

    // Сравниваем результаты CPU и GPU
    bool resultsMatch = true;
    for (int i = 0; i < rows; ++i) {
        if (cpu_symmetryResults[i] != h_symmetryResults[i]) {
            resultsMatch = false;
            break;
        }
    }

    // Выводим результаты сравнения
    if (resultsMatch) {
        std::cout << "Results match between CPU and GPU.\n";
    }
    else {
        std::cout << "Results do not match between CPU and GPU.\n";
    }

    // Выводим процентное ускорение
    float speedup = elapsedTimeCPU * 1000 / elapsedTimeCUDA;

    std::cout << std::endl;
    std::cout << std::setw(20) << std::left << "Measurement" << std::setw(20) << "Time (ms)" << std::endl;
    std::cout << std::setw(20) << std::left << "CPU" << std::setw(20) << elapsedTimeCPU * 1000 << std::endl;
    std::cout << std::setw(20) << std::left << "CUDA" << std::setw(20) << elapsedTimeCUDA << std::endl;
    std::cout << std::endl;
    std::cout << std::setw(20) << std::left << "Speedup" << "x" << speedup << std::endl;


    delete[] h_matrix;
    delete[] h_symmetryResults;


    return 0;
}
